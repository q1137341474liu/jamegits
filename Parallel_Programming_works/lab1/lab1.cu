#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x ;
  if(i<len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;




  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  int size = inputLength * sizeof(float);
  float *A_d, *B_d, *C_d;

  hipMalloc((void **) &A_d, size);
  hipMemcpy(A_d, hostInput1, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &B_d, size);
  hipMemcpy(B_d, hostInput2, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &C_d, size);


  //@@ Copy memory to the GPU here


  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(inputLength/256, 1, 1);
  if (0 != (inputLength % 256)) { DimGrid.x++; }
  dim3 DimBlock(256, 1, 1);


  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd <<<DimGrid, DimBlock>>> (A_d, B_d, C_d, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, C_d, size, hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
